#include "hip/hip_runtime.h"

#include "boruvka.hxx"
#include <hip/hip_runtime.h> // defines threadIdx, blockIdx, blockDim, gridDim
#include <>
#include <gunrock/algorithms/algorithms.hxx> // Gunrock core
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

namespace algos {
struct BoruvkaGunrock::DeviceData {
  using vertex_t = BoruvkaGunrock::vertex_t;
  using edge_t = BoruvkaGunrock::edge_t;
  using weight_t = BoruvkaGunrock::weight_t;

  thrust::device_vector<vertex_t> d_src, d_dst;
  thrust::device_vector<weight_t> d_weight;

  DeviceData() = default;
};

struct BoruvkaGunrock::EdgePair {
  weight_t w;
  edge_t idx;
};

struct BoruvkaGunrock::MinEdgeOp {
  __host__ __device__ EdgePair operator()(EdgePair const &a,
                                          EdgePair const &b) const {
    return (a.w <= b.w) ? a : b;
  }
};

BoruvkaGunrock::BoruvkaGunrock()
    : num_vertices(0), num_edges(0), dev_(new DeviceData()) {}

BoruvkaGunrock::~BoruvkaGunrock() = default;

void BoruvkaGunrock::load_graph(const std::filesystem::path &file_path) {
  using namespace gunrock;
  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  auto loaded = mm.load(file_path);
  auto &coo = std::get<1>(loaded);

  // Define amount edges and verticies
  auto host_rows = coo.row_indices;
  auto host_cols = coo.column_indices;
  auto host_vals = coo.nonzero_values;
  edge_t original_edges = static_cast<edge_t>(host_rows.size());

  // Define max id of the vertex
  vertex_t max_v = 0;
  for (edge_t i = 0; i < original_edges; ++i) {
    max_v = std::max({max_v, host_rows[i], host_cols[i]});
  }
  num_vertices = max_v + 1;

  // Arrays of edges
  thrust::host_vector<vertex_t> h_src;
  thrust::host_vector<vertex_t> h_dst;
  thrust::host_vector<weight_t> h_weight;
  h_src.reserve(2 * original_edges);
  h_dst.reserve(2 * original_edges);
  h_weight.reserve(2 * original_edges);
  for (edge_t i = 0; i < original_edges; ++i) {
    auto u = host_rows[i];
    auto v = host_cols[i];
    auto w = host_vals[i];
    h_src.push_back(u);
    h_dst.push_back(v);
    h_weight.push_back(w);
    if (u != v) {
      h_src.push_back(v);
      h_dst.push_back(u);
      h_weight.push_back(w);
    }
  }
  num_edges = static_cast<edge_t>(h_src.size());

  // Copy to GPU
  dev_->d_src = h_src;
  dev_->d_dst = h_dst;
  dev_->d_weight = h_weight;
}

std::chrono::seconds BoruvkaGunrock::compute() {
  mst_edges.clear();
  auto start = std::chrono::steady_clock::now();
  if (num_vertices == 0)
    return {};

  // Initialize component array: each vertex in its own component (comp[v] = v)
  thrust::device_vector<vertex_t> comp(num_vertices);
  thrust::sequence(comp.begin(), comp.end(), 0); // comp[i] = i

  // Buffers for keys and values when finding minimum edges
  thrust::device_vector<vertex_t> keys(2 * num_edges);
  thrust::device_vector<EdgePair> vals(2 * num_edges);
  thrust::device_vector<vertex_t> comp_keys_out(
      num_vertices); // keys: component IDs
  thrust::device_vector<EdgePair> comp_vals_out(
      num_vertices); // vals: candidate min-edge per component
  bool merged = true;

  // Repeat until no more merges occur
  while (merged) {
    merged = false;
    // Step 1: Record edge candidates into keys/vals
    // Each edge (u,v) adds two entries—one for u’s comp, one for v’s.
    // If both ends share a comp, mark entry invalid (key = -1).
    vertex_t *comp_ptr = thrust::raw_pointer_cast(comp.data());
    vertex_t *src_ptr = thrust::raw_pointer_cast(dev_->d_src.data());
    vertex_t *dst_ptr = thrust::raw_pointer_cast(dev_->d_dst.data());
    weight_t *w_ptr = thrust::raw_pointer_cast(dev_->d_weight.data());
    vertex_t *keys_ptr = thrust::raw_pointer_cast(keys.data());
    EdgePair *vals_ptr = thrust::raw_pointer_cast(vals.data());
    edge_t m = num_edges;

// Launch CUDA kernel: one thread per edge
#pragma omp target teams distribute parallel for is_device_ptr(                \
        comp_ptr, src_ptr, dst_ptr, w_ptr, keys_ptr, vals_ptr)
    for (edge_t e = 0; e < m; ++e) {
      vertex_t u = src_ptr[e];
      vertex_t v = dst_ptr[e];
      weight_t w = w_ptr[e];
      vertex_t comp_u = comp_ptr[u];
      vertex_t comp_v = comp_ptr[v];
      if (comp_u == comp_v) {
        // If edge is internal to a component: mark invalid
        keys_ptr[2 * e] = -1;
        vals_ptr[2 * e] = {w, e};
        keys_ptr[2 * e + 1] = -1;
        vals_ptr[2 * e + 1] = {w, e};
      } else {
        // Otherwise: emit both directions as valid
        keys_ptr[2 * e] = comp_u;
        vals_ptr[2 * e] = {w, e};
        keys_ptr[2 * e + 1] = comp_v;
        vals_ptr[2 * e + 1] = {w, e};
      }
    }

    // Step 2: Filter out invalid entries (key = -1)
    auto zip_begin = thrust::make_zip_iterator(
        thrust::make_tuple(keys.begin(), vals.begin()));
    auto zip_end =
        thrust::make_zip_iterator(thrust::make_tuple(keys.end(), vals.end()));
    auto new_end = thrust::remove_if(
        zip_begin, zip_end,
        [] __host__ __device__(const thrust::tuple<vertex_t, EdgePair> &kv) {
          vertex_t comp_id = thrust::get<0>(kv);
          return comp_id == -1;
        });
    size_t new_size = new_end - zip_begin;
    if (new_size == 0) {
      // If no entries remain, MST is complete.
      break;
    }

    // Resize vectors to new_size after filtering.
    keys.resize(new_size);
    vals.resize(new_size);
    // Step 3: Sort by component ID and pick the minimum edge for each group
    thrust::sort_by_key(keys.begin(), keys.end(), vals.begin());

    // Perform reduce_by_key on (keys, vals) to pick the lightest edge for each
    auto reduce_end = thrust::reduce_by_key(
        keys.begin(), keys.end(), vals.begin(), comp_keys_out.begin(),
        comp_vals_out.begin(), thrust::equal_to<vertex_t>(), MinEdgeOp());
    size_t num_comps_found =
        reduce_end.first -
        comp_keys_out
            .begin(); // comp numFound = number of comps that got an edge

    if (num_comps_found == 0) {
      break; // if numFound == 0: no more cross‐component edges → MST complete
    }

    // Step 4: merge components using those edges
    thrust::device_vector<vertex_t> newComp_map(num_vertices);
    // Initialize newComp_map[i] = i
    thrust::sequence(newComp_map.begin(), newComp_map.end(), 0);
    vertex_t *newcomp_ptr = thrust::raw_pointer_cast(newComp_map.data());
    EdgePair *out_vals_ptr = thrust::raw_pointer_cast(comp_vals_out.data());
    size_t k = num_comps_found;

// CUDA kernel: for each (compID → EdgePair{w,e}):
#pragma omp target teams distribute parallel for is_device_ptr(                \
        comp_ptr, src_ptr, dst_ptr, out_keys_ptr, out_vals_ptr, newcomp_ptr)
    for (size_t i = 0; i < k; ++i) {
      EdgePair ep = out_vals_ptr[i];
      edge_t e = ep.idx;

      // Recover endpoints u,v and their comp labels
      vertex_t u = src_ptr[e];
      vertex_t v = dst_ptr[e];
      vertex_t comp_u = comp_ptr[u];
      vertex_t comp_v = comp_ptr[v];
      if (comp_u == comp_v) {
        continue; // if same label: skip
      }
      // Select the new component representative (smallest ID for determinism)
      vertex_t root = (comp_u < comp_v ? comp_u : comp_v);
      vertex_t other = (comp_u < comp_v ? comp_v : comp_u);

      // Merge: redirect 'other' to 'root'
      newcomp_ptr[other] = root;
    }

    // Flatten union chains with pointer jumping on newComp_map
    bool updated = true;
    int iter = 0;
    while (updated && iter < 10) {
      updated = false;

// Atomic update: newComp_map[x] = newComp_map[newComp_map[x]]
#pragma omp target teams distribute parallel for is_device_ptr(newcomp_ptr)
      for (vertex_t i = 0; i < num_vertices; ++i) {
        vertex_t parent = newcomp_ptr[i];
        vertex_t grandparent = newcomp_ptr[parent];
        if (grandparent != parent) {
          newcomp_ptr[i] = grandparent;
          updated = true;
        }
      }
      iter++;
    }

// Update component labels for all vertices: comp[v] = newComp_map[ comp[v] ]
#pragma omp target teams distribute parallel for is_device_ptr(                \
        comp_ptr, newcomp_ptr)
    for (vertex_t v = 0; v < num_vertices; ++v) {
      comp_ptr[v] = newcomp_ptr[comp_ptr[v]];
    }

    // Step 5: add the chosen edges to the MST result (on host)
    // Copy (weight, edgeIndex) pairs from comp_vals_out and corresponding
    // component IDs from comp_keys_out to host To avoid duplicate edges, use a
    // flag array marking edges already added
    std::vector<vertex_t> out_keys_host(num_comps_found);
    std::vector<EdgePair> out_vals_host(num_comps_found);
    thrust::copy(comp_keys_out.begin(), comp_keys_out.begin() + num_comps_found,
                 out_keys_host.begin());
    thrust::copy(comp_vals_out.begin(), comp_vals_out.begin() + num_comps_found,
                 out_vals_host.begin());

    // Recover the actual vertices of each edge
    static std::vector<char> edge_used;
    edge_used.assign(num_edges, 0);
    for (size_t i = 0; i < num_comps_found; ++i) {
      edge_t e = out_vals_host[i].idx;
      weight_t w = out_vals_host[i].w;

      // After the final comp update, u and v are in the same component
      vertex_t u = (vertex_t)dev_->d_src[e];
      vertex_t v = (vertex_t)dev_->d_dst[e];
      if (!edge_used[e]) {
        edge_used[e] = 1;
        mst_edges.emplace_back(u, v, w);
      }
    }
    merged = true; // if merges occurred, continue the loop
  }
  auto end = std::chrono::steady_clock::now();
  return std::chrono::duration_cast<std::chrono::seconds>(end - start);
}

Tree BoruvkaGunrock::get_result() {
  // Initialize tree structure: parent = -1 for all vertices
  std::vector<int> parent(num_vertices, -1);
  float total_weight = 0.0f;

  // Fill parent array from the MST edge list
  for (auto &e : mst_edges) {
    vertex_t u, v;
    weight_t w;
    std::tie(u, v, w) = e;

    // If v has no parent yet, set u as its parent
    if (parent[v] == -1 && parent[u] != v) {
      parent[v] = u;
    } else if (parent[u] == -1 && parent[v] != u) {
      parent[u] = v;
    }
    total_weight += w;
  }
  return Tree(num_vertices, parent, total_weight);
}
} // namespace algos