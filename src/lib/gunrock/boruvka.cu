#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> // defines threadIdx, blockIdx, blockDim, gridDim
#include <>
#include <gunrock/algorithms/algorithms.hxx> // Gunrock core
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#include "boruvka.hxx"
#include "loader.hxx"

namespace algos {
struct BoruvkaGunrock::DeviceData {
  using vertex_t = BoruvkaGunrock::vertex_t;
  using edge_t = BoruvkaGunrock::edge_t;
  using weight_t = BoruvkaGunrock::weight_t;

  thrust::device_vector<vertex_t> d_src, d_dst;
  thrust::device_vector<weight_t> d_weight;

  DeviceData() = default;
};

struct BoruvkaGunrock::EdgePair {
  weight_t w;
  edge_t idx;
};

struct BoruvkaGunrock::MinEdgeOp {
  __host__ __device__ EdgePair operator()(EdgePair const &a,
                                          EdgePair const &b) const {
    return (a.w <= b.w) ? a : b;
  }
};

// CUDA kernel to record edge candidates into keys/vals
__global__ void
step1_kernel(int m,         // number of edges
             int *comp_ptr, // comp array, length = num_vertices
             int *src_ptr,  // src array,  length = m
             int *dst_ptr,  // dst array,  length = m
             float *w_ptr,  // weight array,length = m
             int *keys_ptr, // keys array, length = 2*m
             BoruvkaGunrock::EdgePair *vals_ptr) // vals array, length = 2*m
{
  int e = blockIdx.x * blockDim.x + threadIdx.x;
  if (e >= m)
    return;

  int u = src_ptr[e];
  int v = dst_ptr[e];
  float w = w_ptr[e];
  int cu = comp_ptr[u];
  int cv = comp_ptr[v];

  if (cu == cv) {
    keys_ptr[2 * e] = -1;
    vals_ptr[2 * e] = {w, e};
    keys_ptr[2 * e + 1] = -1;
    vals_ptr[2 * e + 1] = {w, e};
  } else {
    keys_ptr[2 * e] = cu;
    vals_ptr[2 * e] = {w, e};
    keys_ptr[2 * e + 1] = cv;
    vals_ptr[2 * e + 1] = {w, e};
  }
}

__global__ void step2_kernel(int k, BoruvkaGunrock::vertex_t *src_ptr,
                             BoruvkaGunrock::vertex_t *dst_ptr,
                             BoruvkaGunrock::vertex_t *comp_ptr,
                             BoruvkaGunrock::vertex_t *newcomp_ptr,
                             BoruvkaGunrock::EdgePair *out_vals_ptr) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= k)
    return;

  auto ep = out_vals_ptr[i];
  int e = ep.idx;

  auto u = src_ptr[e];
  auto v = dst_ptr[e];
  auto cu = comp_ptr[u];
  auto cv = comp_ptr[v];
  if (cu == cv)
    return;

  // choose root/other
  auto root = (cu < cv ? cu : cv);
  auto other = (cu < cv ? cv : cu);
  // merge
  newcomp_ptr[other] = root;
}

__global__ void pointer_jump_kernel(int num_vertices,
                                    BoruvkaGunrock::vertex_t *newcomp_ptr) {
  int v = blockIdx.x * blockDim.x + threadIdx.x;
  if (v >= num_vertices)
    return;
  auto parent = newcomp_ptr[v];
  auto grandparent = newcomp_ptr[parent];
  if (grandparent != parent) {
    newcomp_ptr[v] = grandparent;
  }
}

__global__ void update_comp_kernel(int num_vertices,
                                   BoruvkaGunrock::vertex_t *comp_ptr,
                                   BoruvkaGunrock::vertex_t *newcomp_ptr) {
  int v = blockIdx.x * blockDim.x + threadIdx.x;
  if (v >= num_vertices)
    return;
  comp_ptr[v] = newcomp_ptr[comp_ptr[v]];
}

BoruvkaGunrock::BoruvkaGunrock()
    : num_vertices(0), num_edges(0), dev_(new DeviceData()) {}

BoruvkaGunrock::~BoruvkaGunrock() = default;

void BoruvkaGunrock::load_graph(const std::filesystem::path &file_path) {
  std::vector<vertex_t> host_rows, host_cols;
  std::vector<weight_t> host_vals;
  detail::load_mtx_coo<vertex_t, edge_t, weight_t>(file_path, host_rows,
                                                   host_cols, host_vals);
  edge_t original_edges = static_cast<edge_t>(host_rows.size());

  // 2) Compute number of vertices = max index + 1
  vertex_t max_v = 0;
  for (edge_t i = 0; i < original_edges; ++i) {
    max_v = std::max({max_v, host_rows[i], host_cols[i]});
  }
  num_vertices = max_v + 1;

  // 3) Build undirected edge lists (duplicate if u!=v)
  thrust::host_vector<vertex_t> h_src, h_dst;
  thrust::host_vector<weight_t> h_w;
  h_src.reserve(2 * original_edges);
  h_dst.reserve(2 * original_edges);
  h_w.reserve(2 * original_edges);

  for (edge_t i = 0; i < original_edges; ++i) {
    auto u = host_rows[i];
    auto v = host_cols[i];
    auto w = host_vals[i];
    h_src.push_back(u);
    h_dst.push_back(v);
    h_w.push_back(w);
    if (u != v) {
      h_src.push_back(v);
      h_dst.push_back(u);
      h_w.push_back(w);
    }
  }
  num_edges = static_cast<edge_t>(h_src.size());

  // 4) Copy into your device‐side vectors
  dev_->d_src = h_src;
  dev_->d_dst = h_dst;
  dev_->d_weight = h_w;
}

std::chrono::milliseconds BoruvkaGunrock::compute() {
  mst_edges.clear();
  auto start = std::chrono::steady_clock::now();
  if (num_vertices == 0)
    return {};

  // Initialize component array: each vertex in its own component (comp[v] = v)
  thrust::device_vector<vertex_t> comp(num_vertices);
  thrust::sequence(comp.begin(), comp.end(), 0); // comp[i] = i

  // Buffers for keys and values when finding minimum edges
  thrust::device_vector<vertex_t> keys(2 * num_edges);
  thrust::device_vector<EdgePair> vals(2 * num_edges);
  thrust::device_vector<vertex_t> comp_keys_out(
      num_vertices); // keys: component IDs
  thrust::device_vector<EdgePair> comp_vals_out(
      num_vertices); // vals: candidate min-edge per component
  edge_t m0 = num_edges / 2;
  std::vector<char> edge_used(m0, 0);
  edge_used.assign(m0, 0);
  bool merged = true;

  // Repeat until no more merges occur
  while (merged) {
    merged = false;
    keys.resize(2 * num_edges);
    vals.resize(2 * num_edges);
    // Step 1: Record edge candidates into keys/vals
    // Each edge (u,v) adds two entries—one for u’s comp, one for v’s.
    // If both ends share a comp, mark entry invalid (key = -1).
    vertex_t *comp_ptr = thrust::raw_pointer_cast(comp.data());
    vertex_t *src_ptr = thrust::raw_pointer_cast(dev_->d_src.data());
    vertex_t *dst_ptr = thrust::raw_pointer_cast(dev_->d_dst.data());
    weight_t *w_ptr = thrust::raw_pointer_cast(dev_->d_weight.data());
    vertex_t *keys_ptr = thrust::raw_pointer_cast(keys.data());
    EdgePair *vals_ptr = thrust::raw_pointer_cast(vals.data());
    edge_t m = num_edges;

    // Launch CUDA kernel: one thread per edge
    {
      int threads = 256;
      int blocks = (m + threads - 1) / threads;
      step1_kernel<<<blocks, threads>>>(m, comp_ptr, src_ptr, dst_ptr, w_ptr,
                                        keys_ptr, vals_ptr);
      hipDeviceSynchronize();
    }

    // Step 2: Filter out invalid entries (key = -1)
    auto zip_begin = thrust::make_zip_iterator(
        thrust::make_tuple(keys.begin(), vals.begin()));
    auto zip_end =
        thrust::make_zip_iterator(thrust::make_tuple(keys.end(), vals.end()));
    auto new_end = thrust::remove_if(
        zip_begin, zip_end,
        [] __host__ __device__(const thrust::tuple<vertex_t, EdgePair> &kv) {
          vertex_t comp_id = thrust::get<0>(kv);
          return comp_id == -1;
        });
    size_t new_size = new_end - zip_begin;
    if (new_size == 0)
      // If no entries remain, MST is complete.
      break;

    // Resize vectors to new_size after filtering.
    keys.resize(new_size);
    vals.resize(new_size);
    // Step 3: Sort by component ID and pick the minimum edge for each group
    thrust::sort_by_key(keys.begin(), keys.end(), vals.begin());

    // Perform reduce_by_key on (keys, vals) to pick the lightest edge for each
    auto reduce_end = thrust::reduce_by_key(
        keys.begin(), keys.end(), vals.begin(), comp_keys_out.begin(),
        comp_vals_out.begin(), thrust::equal_to<vertex_t>(), MinEdgeOp());
    size_t num_comps_found =
        reduce_end.first -
        comp_keys_out
            .begin(); // comp numFound = number of comps that got an edge

    if (num_comps_found == 0)
      break; // if numFound == 0: no more cross‐component edges → MST complete

    // Step 4: merge components using those edges
    thrust::device_vector<vertex_t> newComp_map(num_vertices);
    // Initialize newComp_map[i] = i
    thrust::sequence(newComp_map.begin(), newComp_map.end(), 0);
    vertex_t *newcomp_ptr = thrust::raw_pointer_cast(newComp_map.data());
    EdgePair *out_vals_ptr = thrust::raw_pointer_cast(comp_vals_out.data());
    size_t k = num_comps_found;

    // CUDA kernel: for each (compID → EdgePair{w,e}):
    {
      int threads = 256;
      int blocks = (k + threads - 1) / threads;
      step2_kernel<<<blocks, threads>>>(k, src_ptr, dst_ptr, comp_ptr,
                                        newcomp_ptr, out_vals_ptr);
      hipDeviceSynchronize();
    }

    // Flatten union chains with pointer jumping on newComp_map
    bool updated = true;
    int iter = 0;
    while (updated && iter < 10) {
      updated = false;

      // Atomic update: newComp_map[x] = newComp_map[newComp_map[x]]
      {
        int threads = 256;
        int blocks = (num_vertices + threads - 1) / threads;
        for (int iter = 0; iter < 10; ++iter) {
          pointer_jump_kernel<<<blocks, threads>>>(num_vertices, newcomp_ptr);
          hipDeviceSynchronize();
        }
      }
    }

    // Update component labels for all vertices: comp[v] = newComp_map[
    // comp[v] ]
    {
      int threads = 256;
      int blocks = (num_vertices + threads - 1) / threads;
      update_comp_kernel<<<blocks, threads>>>(num_vertices, comp_ptr,
                                              newcomp_ptr);
      hipDeviceSynchronize();
    }

    // Step 5: add the chosen edges to the MST result (on host)
    // Copy (weight, edgeIndex) pairs from comp_vals_out and corresponding
    // component IDs from comp_keys_out to host To avoid duplicate edges, use
    // a flag array marking edges already added
    std::vector<vertex_t> out_keys_host(num_comps_found);
    std::vector<EdgePair> out_vals_host(num_comps_found);
    thrust::copy(comp_keys_out.begin(), comp_keys_out.begin() + num_comps_found,
                 out_keys_host.begin());
    thrust::copy(comp_vals_out.begin(), comp_vals_out.begin() + num_comps_found,
                 out_vals_host.begin());

    // Recover the actual vertices of each edge
    for (size_t i = 0; i < num_comps_found; ++i) {
      edge_t e = out_vals_host[i].idx;
      edge_t orig = e / 2; // индекс неориентированного ребра
      weight_t w = out_vals_host[i].w;
      if (!edge_used[orig]) {
        edge_used[orig] = 1;
        vertex_t u = dev_->d_src[e];
        vertex_t v = dev_->d_dst[e];
        mst_edges.emplace_back(u, v, w);
      }
    }
    merged = true; // if merges occurred, continue the loop
  }
  auto end = std::chrono::steady_clock::now();
  return std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
}

Tree BoruvkaGunrock::get_result() {
  // Initialize tree structure: parent = -1 for all vertices
  std::vector<int> parent(num_vertices, -1);
  float total_weight = 0.0f;

  // Fill parent array from the MST edge list
  for (auto &e : mst_edges) {
    vertex_t u, v;
    weight_t w;
    std::tie(u, v, w) = e;

    // If v has no parent yet, set u as its parent
    if (parent[v] == -1 && parent[u] != v) {
      parent[v] = u;
    } else if (parent[u] == -1 && parent[v] != u) {
      parent[u] = v;
    }
    total_weight += w;
  }
  return Tree(num_vertices, parent, total_weight);
}
} // namespace algos