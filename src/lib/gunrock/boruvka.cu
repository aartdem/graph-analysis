
#include <hip/hip_runtime.h>

// #include "boruvka.hxx"

// namespace algos {
// // Загрузка графа из файла в формате Matrix Market (MTX)
// void BoruvkaGunrock::load_graph(const std::filesystem::path &file_path) {
//   using namespace gunrock;
//   io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
//   auto loaded = mm.load(file_path);
//   auto &coo = std::get<1>(loaded);

//   // Определяем количество рёбер и вершин
//   auto host_rows = coo.row_indices;
//   auto host_cols = coo.column_indices;
//   auto host_vals = coo.nonzero_values;
//   edge_t original_edges = static_cast<edge_t>(host_rows.size());
//   // Определяем максимальный идентификатор вершины
//   vertex_t max_v = 0;
//   for (edge_t i = 0; i < original_edges; ++i) {
//     max_v = std::max({max_v, host_rows[i], host_cols[i]});
//   }
//   num_vertices = max_v + 1;

//   // Подготавливаем массивы ребер (неориентированный, дублируем)
//   thrust::host_vector<vertex_t> h_src;
//   thrust::host_vector<vertex_t> h_dst;
//   thrust::host_vector<weight_t> h_weight;
//   h_src.reserve(2 * original_edges);
//   h_dst.reserve(2 * original_edges);
//   h_weight.reserve(2 * original_edges);

//   for (edge_t i = 0; i < original_edges; ++i) {
//     auto u = host_rows[i];
//     auto v = host_cols[i];
//     auto w = host_vals[i];
//     h_src.push_back(u);
//     h_dst.push_back(v);
//     h_weight.push_back(w);
//     if (u != v) {
//       h_src.push_back(v);
//       h_dst.push_back(u);
//       h_weight.push_back(w);
//     }
//   }
//   num_edges = static_cast<edge_t>(h_src.size());

//   // Копируем на GPU
//   d_src = h_src;
//   d_dst = h_dst;
//   d_weight = h_weight;
// }

// // Основной метод вычисления MST
// std::chrono::seconds BoruvkaGunrock::compute() {
//   mst_edges.clear();
//   auto start = std::chrono::steady_clock::now();
//   if (num_vertices == 0)
//     return {};
//   // Инициализация массива компонент: в начале каждая вершина в своей
//   // компоненте (comp[v] = v)
//   thrust::device_vector<vertex_t> comp(num_vertices);
//   thrust::sequence(comp.begin(), comp.end(), 0); // comp[i] = i

//   // Массивы-буферы для ключей и значений при поиске минимальных ребер
//   thrust::device_vector<vertex_t> keys(2 * num_edges);
//   thrust::device_vector<EdgePair> vals(2 * num_edges);
//   thrust::device_vector<vertex_t> comp_keys_out(
//       num_vertices); // выходные ключи (компоненты)
//   thrust::device_vector<EdgePair> comp_vals_out(
//       num_vertices); // выходные значения (мин.ребро для компоненты)

//   bool merged = true;
//   // Выполняем итерации, пока происходит объединение компонент
//   while (merged) {
//     merged = false;
//     // Шаг 1: Для каждого ребра записываем кандидаты в массивы keys/vals.
//     // Каждый ребро (u,v) дает два кандидата: для компоненты u и для
//     // компоненты v. Если вершины в одной компоненте, помечаем кандидатов как
//     // невалидных (ключ = -1).
//     vertex_t *comp_ptr = thrust::raw_pointer_cast(comp.data());
//     vertex_t *src_ptr = thrust::raw_pointer_cast(d_src.data());
//     vertex_t *dst_ptr = thrust::raw_pointer_cast(d_dst.data());
//     weight_t *w_ptr = thrust::raw_pointer_cast(d_weight.data());
//     vertex_t *keys_ptr = thrust::raw_pointer_cast(keys.data());
//     EdgePair *vals_ptr = thrust::raw_pointer_cast(vals.data());
//     edge_t m = num_edges;
// // Запускаем CUDA-ядро одной нитью на ребро:
// // (В реальном коде следует проверить CUDA ошибки, для краткости опущено)
// #pragma omp target teams distribute parallel for is_device_ptr(                \
//         comp_ptr, src_ptr, dst_ptr, w_ptr, keys_ptr, vals_ptr)
//     for (edge_t e = 0; e < m; ++e) {
//       vertex_t u = src_ptr[e];
//       vertex_t v = dst_ptr[e];
//       weight_t w = w_ptr[e];
//       vertex_t comp_u = comp_ptr[u];
//       vertex_t comp_v = comp_ptr[v];
//       if (comp_u == comp_v) {
//         // Ребро внутри одной компоненты — помечаем как невалидное
//         keys_ptr[2 * e] = -1;
//         vals_ptr[2 * e] = {w, e};
//         keys_ptr[2 * e + 1] = -1;
//         vals_ptr[2 * e + 1] = {w, e};
//       } else {
//         // Ребро между разными компонентами: добавляем два направления
//         keys_ptr[2 * e] = comp_u;
//         vals_ptr[2 * e] = {w, e};
//         keys_ptr[2 * e + 1] = comp_v;
//         vals_ptr[2 * e + 1] = {w, e};
//       }
//     }
//     // Шаг 2: Фильтрация невалидных записей (где ключ = -1)
//     auto zip_begin = thrust::make_zip_iterator(
//         thrust::make_tuple(keys.begin(), vals.begin()));
//     auto zip_end =
//         thrust::make_zip_iterator(thrust::make_tuple(keys.end(), vals.end()));
//     auto new_end = thrust::remove_if(
//         zip_begin, zip_end,
//         [] __host__ __device__(const thrust::tuple<vertex_t, EdgePair> &kv) {
//           vertex_t comp_id = thrust::get<0>(kv);
//           return comp_id == -1;
//         });
//     size_t new_size = new_end - zip_begin;
//     if (new_size == 0) {
//       // Нет ребер между компонентами – MST построено (либо несколько
//       // изолированных компонент)
//       break;
//     }
//     // Обрезаем векторы до new_size после удаления невалидных элементов
//     keys.resize(new_size);
//     vals.resize(new_size);
//     // Шаг 3: Группировка по компонентам и выбор минимального ребра для каждой
//     // компоненты Сортируем по ключам-компонентам, чтобы одинаковые компоненты
//     // шли подряд
//     thrust::sort_by_key(keys.begin(), keys.end(), vals.begin());
//     // Выполняем reduce_by_key для нахождения минимального ребра по весу для
//     // каждой группы ключей
//     auto reduce_end = thrust::reduce_by_key(
//         keys.begin(), keys.end(), vals.begin(), comp_keys_out.begin(),
//         comp_vals_out.begin(), thrust::equal_to<vertex_t>(), MinEdgeOp());
//     size_t num_comps_found =
//         reduce_end.first -
//         comp_keys_out.begin(); // число компонент, для которых найдено ребро
//     if (num_comps_found == 0) {
//       break; // не найдено ни одного внешнего ребра (на случай неполной связи)
//     }
//     // Шаг 4: Объединение компонент по выбранным ребрам.
//     // Для каждой записи (comp_id -> EdgePair{w, e}) из результата:
//     // определяем соседнюю компоненту и выполняем слияние (переназначение
//     // меток). newComp_map[x] будет новой меткой для компоненты с
//     // идентификатором x после объединения.
//     thrust::device_vector<vertex_t> newComp_map(num_vertices);
//     // Изначально newComp_map[i] = i (каждая компонента остается сама собой,
//     // если не будет переопределена)
//     thrust::sequence(newComp_map.begin(), newComp_map.end(), 0);
//     vertex_t *newcomp_ptr = thrust::raw_pointer_cast(newComp_map.data());
//     vertex_t *out_keys_ptr = thrust::raw_pointer_cast(comp_keys_out.data());
//     EdgePair *out_vals_ptr = thrust::raw_pointer_cast(comp_vals_out.data());
//     size_t k = num_comps_found;
// // Ядро: объединяем компоненты (связываем "старшую" компоненту с "младшей")
// #pragma omp target teams distribute parallel for is_device_ptr(                \
//         comp_ptr, src_ptr, dst_ptr, out_keys_ptr, out_vals_ptr, newcomp_ptr)
//     for (size_t i = 0; i < k; ++i) {
//       vertex_t comp_id = out_keys_ptr[i];
//       EdgePair ep = out_vals_ptr[i];
//       edge_t e = ep.idx;
//       // Восстанавливаем концы ребра
//       vertex_t u = src_ptr[e];
//       vertex_t v = dst_ptr[e];
//       // Определяем метки компонентов концов ребра (могут совпадать с comp_id
//       // или быть другой стороной)
//       vertex_t comp_u = comp_ptr[u];
//       vertex_t comp_v = comp_ptr[v];
//       if (comp_u == comp_v) {
//         continue; // обе вершины уже в одной компоненте
//       }
//       // Выбираем нового представителя компоненты (минимальный id для
//       // детерминизма)
//       vertex_t root = (comp_u < comp_v ? comp_u : comp_v);
//       vertex_t other = (comp_u < comp_v ? comp_v : comp_u);
//       // Объединяем: перенаправляем 'other' к 'root'
//       newcomp_ptr[other] = root;
//     }
//     // Сжимаем цепочки объединения (pointer jumping для newComp_map)
//     bool updated = true;
//     int iter = 0;
//     while (updated && iter < 10) { // максимум 10 итераций для безопасности
//       updated = false;
// // Одновременное обновление: newComp_map[x] = newComp_map[newComp_map[x]]
// #pragma omp target teams distribute parallel for is_device_ptr(newcomp_ptr)
//       for (vertex_t i = 0; i < num_vertices; ++i) {
//         vertex_t parent = newcomp_ptr[i];
//         vertex_t grandparent = newcomp_ptr[parent];
//         if (grandparent != parent) {
//           newcomp_ptr[i] = grandparent;
//           updated = true;
//         }
//       }
//       iter++;
//     }
// // Обновляем метки компонент для всех вершин: comp[v] = newComp_map[ comp[v] ]
// #pragma omp target teams distribute parallel for is_device_ptr(                \
//         comp_ptr, newcomp_ptr)
//     for (vertex_t v = 0; v < num_vertices; ++v) {
//       comp_ptr[v] = newcomp_ptr[comp_ptr[v]];
//     }
//     // Шаг 5: Добавление выбранных ребер в результат MST (на хосте).
//     // Копируем пары (вес, индекс ребра) из comp_vals_out и соответствующие
//     // компоненты из comp_keys_out на хост.
//     std::vector<vertex_t> out_keys_host(num_comps_found);
//     std::vector<EdgePair> out_vals_host(num_comps_found);
//     thrust::copy(comp_keys_out.begin(), comp_keys_out.begin() + num_comps_found,
//                  out_keys_host.begin());
//     thrust::copy(comp_vals_out.begin(), comp_vals_out.begin() + num_comps_found,
//                  out_vals_host.begin());
//     // Чтобы избежать дублирования одного и того же ребра дважды, используем
//     // массив пометок уже добавленных ребер.
//     static std::vector<char>
//         edge_used; // статический, чтобы не перераспределять каждый раз (можно
//                    // и как поле класса)
//     edge_used.assign(num_edges, 0);
//     for (size_t i = 0; i < num_comps_found; ++i) {
//       edge_t e = out_vals_host[i].idx;
//       weight_t w = out_vals_host[i].w;
//       // Найдем фактические вершины этого ребра
//       // (После последнего обновления comp, вершины u и v уже в одной
//       // компоненте)
//       vertex_t u = (vertex_t)d_src[e];
//       vertex_t v = (vertex_t)d_dst[e];
//       if (!edge_used[e]) {
//         edge_used[e] = 1;
//         mst_edges.emplace_back(u, v, w);
//       }
//     }
//     merged = true; // произошли объединения, продолжаем цикл
//   } // конец while

//   auto end = std::chrono::steady_clock::now();
//   return std::chrono::duration_cast<std::chrono::seconds>(end - start);
// }

// Tree BoruvkaGunrock::get_result() {
//   // Инициализируем дерево: parent = -1 для всех вершин
//   std::vector<int> parent(num_vertices, -1);
//   float total_weight = 0.0f;
//   // Заполняем parent по списку рёбер MST
//   for (auto &e : mst_edges) {
//     vertex_t u, v;
//     weight_t w;
//     std::tie(u, v, w) = e;
//     // Если v еще без родителя, делаем u его родителем
//     if (parent[v] == -1 && parent[u] != v) {
//       parent[v] = u;
//     } else if (parent[u] == -1 && parent[v] != u) {
//       parent[u] = v;
//     }
//     total_weight += w;
//   }
//   return Tree(num_vertices, parent, total_weight);
// }
// } // namespace algos