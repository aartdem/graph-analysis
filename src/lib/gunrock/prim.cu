#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>

#include "loader.hxx"
#include "prim.hxx"

namespace algos {

// Structure holding the graph in CSR format and auxiliary GPU buffers
struct PrimGunrock::DeviceData {
  using vertex_t = PrimGunrock::vertex_t;
  using edge_t = PrimGunrock::edge_t;
  using weight_t = PrimGunrock::weight_t;

  // CSR representation
  thrust::device_vector<edge_t> d_row_offsets;   // length = num_vertices + 1
  thrust::device_vector<vertex_t> d_col_indices; // length = num_edges
  thrust::device_vector<weight_t> d_weight;      // length = num_edges

  // Host copy of row_offsets for host-side access
  thrust::host_vector<edge_t> h_row_offsets; // length = num_vertices + 1

  // Buffers for Prim's algorithm
  thrust::device_vector<weight_t> d_key;    // minimum edge weights
  thrust::device_vector<vertex_t> d_parent; // MST parent pointers
  thrust::device_vector<char> d_inMST;      // flags marking inclusion in MST

  DeviceData() = default;
};

PrimGunrock::PrimGunrock()
    : num_vertices(0), num_edges(0), dev_(new DeviceData()) {}

PrimGunrock::~PrimGunrock() = default;

void PrimGunrock::load_graph(const std::filesystem::path &file_path) {
  using vertex_t = PrimGunrock::vertex_t;
  using edge_t = PrimGunrock::edge_t;
  using weight_t = PrimGunrock::weight_t;

  // Load the graph in COO format
  std::vector<vertex_t> coo_row, coo_col;
  std::vector<weight_t> coo_val;
  detail::load_mtx_coo<vertex_t, edge_t, weight_t>(file_path, coo_row, coo_col,
                                                   coo_val);

  // Determine number of vertices and original edges
  edge_t original_edges = static_cast<edge_t>(coo_row.size());
  vertex_t max_v = 0;
  for (vertex_t u : coo_row)
    max_v = std::max(max_v, u);
  for (vertex_t v : coo_col)
    max_v = std::max(max_v, v);
  num_vertices = max_v + 1;

  // Build a symmetric COO (undirected graph)
  std::vector<vertex_t> srcs, dsts;
  std::vector<weight_t> weights;
  srcs.reserve(2 * original_edges);
  dsts.reserve(2 * original_edges);
  weights.reserve(2 * original_edges);

  for (edge_t i = 0; i < original_edges; ++i) {
    vertex_t u = coo_row[i], v = coo_col[i];
    weight_t w = coo_val[i];
    srcs.push_back(u);
    dsts.push_back(v);
    weights.push_back(w);
    if (u != v) {
      srcs.push_back(v);
      dsts.push_back(u);
      weights.push_back(w);
    }
  }
  num_edges = static_cast<edge_t>(srcs.size());

  // Build CSR structure locally
  // Compute vertex degrees
  std::vector<vertex_t> degrees(num_vertices, 0);
  for (edge_t i = 0; i < num_edges; ++i) {
    degrees[srcs[i]]++;
  }

  // Compute exclusive prefix sum for row offsets
  std::vector<vertex_t> row_offsets_local(num_vertices + 1);
  row_offsets_local[0] = 0;
  for (vertex_t i = 0; i < num_vertices; ++i)
    row_offsets_local[i + 1] = row_offsets_local[i] + degrees[i];

  // Initialize cursors and fill column and weight arrays
  std::vector<edge_t> cursor(row_offsets_local.begin(),
                             row_offsets_local.end());
  std::vector<vertex_t> col_idx_local(num_edges);
  std::vector<weight_t> w_local(num_edges);

  for (edge_t i = 0; i < num_edges; ++i) {
    vertex_t u = srcs[i];
    edge_t pos = cursor[u]++;
    col_idx_local[pos] = dsts[i];
    w_local[pos] = weights[i];
  }
  dev_->h_row_offsets = thrust::host_vector<vertex_t>(row_offsets_local.begin(),
                                                      row_offsets_local.end());

  // Copy CSR data to device vectors
  dev_->d_row_offsets = thrust::device_vector<vertex_t>(
      row_offsets_local.begin(), row_offsets_local.end());
  dev_->d_col_indices = thrust::device_vector<vertex_t>(col_idx_local.begin(),
                                                        col_idx_local.end());
  dev_->d_weight =
      thrust::device_vector<weight_t>(w_local.begin(), w_local.end());
}

std::chrono::seconds PrimGunrock::compute() {
  mst_edges.clear();
  using vertex_t = PrimGunrock::vertex_t;
  using edge_t = PrimGunrock::edge_t;
  using weight_t = PrimGunrock::weight_t;
  const weight_t INF = std::numeric_limits<weight_t>::max();

  auto start = std::chrono::steady_clock::now();
  if (num_vertices == 0)
    return {};

  auto &D = *dev_;
  // Resize buffers for Prim's algorithm
  D.d_key.resize(num_vertices);
  D.d_parent.resize(num_vertices);
  D.d_inMST.resize(num_vertices);

  thrust::fill(D.d_key.begin(), D.d_key.end(), INF);
  thrust::fill(D.d_parent.begin(), D.d_parent.end(), -1);
  thrust::fill(D.d_inMST.begin(), D.d_inMST.end(), 0);

  // Start from vertex 0
  D.d_key[0] = 0;
  D.d_parent[0] = 0;

  // Raw pointers for device data
  auto row_ptr = thrust::raw_pointer_cast(D.d_row_offsets.data());
  auto col_ptr = thrust::raw_pointer_cast(D.d_col_indices.data());
  auto w_ptr = thrust::raw_pointer_cast(D.d_weight.data());
  auto key_ptr = thrust::raw_pointer_cast(D.d_key.data());
  auto parent_ptr = thrust::raw_pointer_cast(D.d_parent.data());
  auto inMST_ptr = thrust::raw_pointer_cast(D.d_inMST.data());

  // Main Prim's loop
  for (size_t iter = 0; iter < num_vertices; ++iter) {
    // Select the vertex with the minimum key
    auto it =
        thrust::min_element(thrust::device, D.d_key.begin(), D.d_key.end());
    vertex_t u = it - D.d_key.begin();
    weight_t minKey = *it;
    if (minKey == INF) {
      // Handle disconnected components by resetting a new start vertex
      bool found = false;
      for (vertex_t x = 0; x < num_vertices; ++x) {
        if (D.d_inMST[x] == 0) {
          D.d_key[x] = 0;
          D.d_parent[x] = x;
          it = D.d_key.begin() + x;
          u = x;
          minKey = 0;
          found = true;
          break;
        }
      }
      if (!found)
        break;
    }

    // Include vertex u in the MST
    D.d_inMST[u] = 1;
    *it = INF;
    if (u != D.d_parent[u])
      mst_edges.emplace_back(D.d_parent[u], u, minKey);

    // Update keys of adjacent vertices in parallel
    edge_t e_start = D.h_row_offsets[u];
    edge_t e_end = D.h_row_offsets[u + 1];
    thrust::for_each(thrust::device,
                     thrust::make_counting_iterator<edge_t>(e_start),
                     thrust::make_counting_iterator<edge_t>(e_end),
                     [=] __device__(edge_t e) {
                       vertex_t v = col_ptr[e];
                       weight_t w = w_ptr[e];
                       if (!inMST_ptr[v] && w < key_ptr[v]) {
                         key_ptr[v] = w;
                         parent_ptr[v] = u;
                       }
                     });
  }

  auto end = std::chrono::steady_clock::now();
  return std::chrono::duration_cast<std::chrono::seconds>(end - start);
}

Tree PrimGunrock::get_result() {
  // Copy parent array back to host
  std::vector<int> parent(num_vertices);
  thrust::copy(dev_->d_parent.begin(), dev_->d_parent.end(), parent.begin());

  // Compute total weight of the MST
  float totalWeight = 0;
  for (auto &edge : mst_edges)
    totalWeight += std::get<2>(edge);
  return Tree(num_vertices, parent, totalWeight);
}

} // namespace algos
