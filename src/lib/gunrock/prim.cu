#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>

#include "loader.hxx"
#include "prim.hxx"

namespace algos {

// Structure holding the graph in CSR format and auxiliary GPU buffers
struct PrimGunrock::DeviceData {
  using vertex_t = PrimGunrock::vertex_t;
  using edge_t = PrimGunrock::edge_t;
  using weight_t = PrimGunrock::weight_t;

  thrust::device_vector<edge_t> d_row_offsets;
  thrust::device_vector<vertex_t> d_col_indices;
  thrust::device_vector<weight_t> d_weight;

  thrust::device_vector<weight_t> d_key;
  thrust::device_vector<vertex_t> d_parent;
  thrust::device_vector<char> d_inMST;

  // buffers for block reduction
  thrust::device_vector<weight_t> d_min_key;
  thrust::device_vector<vertex_t> d_min_idx;
};

// Block-level reduction kernel: find min(key) and its index in each block
__global__ void minKeyReduce(const PrimGunrock::weight_t *keys,
                             PrimGunrock::weight_t *block_min_key,
                             PrimGunrock::vertex_t *block_min_idx, int n,
                             PrimGunrock::weight_t INF) {
  extern __shared__ char smem[];
  auto *s_keys = reinterpret_cast<PrimGunrock::weight_t *>(smem);
  auto *s_idx = reinterpret_cast<PrimGunrock::vertex_t *>(&s_keys[blockDim.x]);

  int tid = threadIdx.x;
  int gid = blockIdx.x * blockDim.x + tid;

  // initialize
  PrimGunrock::weight_t v = (gid < n) ? keys[gid] : INF;
  PrimGunrock::vertex_t idx = (gid < n) ? gid : -1;
  s_keys[tid] = v;
  s_idx[tid] = idx;
  __syncthreads();

  // tree-based reduction
  for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
    if (tid < offset) {
      if (s_keys[tid + offset] < s_keys[tid]) {
        s_keys[tid] = s_keys[tid + offset];
        s_idx[tid] = s_idx[tid + offset];
      }
    }
    __syncthreads();
  }

  if (tid == 0) {
    block_min_key[blockIdx.x] = s_keys[0];
    block_min_idx[blockIdx.x] = s_idx[0];
  }
}

// Kernel to relax edges of a single vertex u
__global__ void relaxEdges(const PrimGunrock::edge_t *row_offsets,
                           const PrimGunrock::vertex_t *col_indices,
                           const PrimGunrock::weight_t *weights,
                           PrimGunrock::weight_t *keys,
                           PrimGunrock::vertex_t *parents, const char *inMST,
                           PrimGunrock::vertex_t u) {
  int lane = threadIdx.x;
  auto start = row_offsets[u];
  auto end = row_offsets[u + 1];
  for (auto e = start + lane; e < end; e += blockDim.x) {
    auto v = col_indices[e];
    auto w = weights[e];
    if (!inMST[v] && w < keys[v]) {
      keys[v] = w;
      parents[v] = u;
    }
  }
}

PrimGunrock::PrimGunrock()
    : num_vertices(0), num_edges(0), dev_(new DeviceData()) {}

PrimGunrock::~PrimGunrock() = default;

void PrimGunrock::load_graph(const std::filesystem::path &file_path) {
  using vertex_t = PrimGunrock::vertex_t;
  using edge_t = PrimGunrock::edge_t;
  using weight_t = PrimGunrock::weight_t;

  std::vector<vertex_t> coo_row, coo_col;
  std::vector<weight_t> coo_val;
  detail::load_mtx_coo<vertex_t, edge_t, weight_t>(file_path, coo_row, coo_col,
                                                   coo_val);

  edge_t orig_e = (edge_t)coo_row.size();
  vertex_t max_v = 0;
  for (auto u : coo_row)
    max_v = max(max_v, u);
  for (auto v : coo_col)
    max_v = max(max_v, v);
  num_vertices = max_v + 1;

  std::vector<vertex_t> src, dst;
  std::vector<weight_t> wts;
  src.reserve(2 * orig_e);
  dst.reserve(2 * orig_e);
  wts.reserve(2 * orig_e);
  for (edge_t i = 0; i < orig_e; i++) {
    auto u = coo_row[i], v = coo_col[i];
    auto w = coo_val[i];
    src.push_back(u);
    dst.push_back(v);
    wts.push_back(w);
    if (u != v) {
      src.push_back(v);
      dst.push_back(u);
      wts.push_back(w);
    }
  }
  num_edges = (edge_t)src.size();

  std::vector<edge_t> degrees(num_vertices, 0);
  for (edge_t i = 0; i < num_edges; i++)
    degrees[src[i]]++;

  std::vector<edge_t> row_off(num_vertices + 1);
  row_off[0] = 0;
  for (vertex_t i = 0; i < num_vertices; i++)
    row_off[i + 1] = row_off[i] + degrees[i];

  std::vector<edge_t> cursor = row_off;
  std::vector<vertex_t> col_idx(num_edges);
  std::vector<weight_t> w_local(num_edges);
  for (edge_t i = 0; i < num_edges; i++) {
    auto u = src[i];
    auto pos = cursor[u]++;
    col_idx[pos] = dst[i];
    w_local[pos] = wts[i];
  }

  // Copy CSR to device
  auto &D = *dev_;
  D.d_row_offsets = row_off;
  D.d_col_indices = col_idx;
  D.d_weight = w_local;

  // Pre-allocate buffers
  D.d_key.resize(num_vertices);
  D.d_parent.resize(num_vertices);
  D.d_inMST.resize(num_vertices);

  int threads = 256;
  int blocks = (num_vertices + threads - 1) / threads;
  D.d_min_key.resize(blocks);
  D.d_min_idx.resize(blocks);
}

std::chrono::milliseconds PrimGunrock::compute() {
  using weight_t = PrimGunrock::weight_t;
  const weight_t INF = std::numeric_limits<weight_t>::max();
  auto start = std::chrono::steady_clock::now();
  if (num_vertices == 0)
    return {};
  auto &D = *dev_;

  // init
  thrust::fill(D.d_key.begin(), D.d_key.end(), INF);
  thrust::fill(D.d_parent.begin(), D.d_parent.end(), -1);
  thrust::fill(D.d_inMST.begin(), D.d_inMST.end(), 0);
  D.d_key[0] = 0;
  D.d_parent[0] = 0;

  // raw pointers
  auto row_ptr = thrust::raw_pointer_cast(D.d_row_offsets.data());
  auto col_ptr = thrust::raw_pointer_cast(D.d_col_indices.data());
  auto w_ptr = thrust::raw_pointer_cast(D.d_weight.data());
  auto key_ptr = thrust::raw_pointer_cast(D.d_key.data());
  auto p_ptr = thrust::raw_pointer_cast(D.d_parent.data());
  auto in_ptr = thrust::raw_pointer_cast(D.d_inMST.data());
  auto bk_ptr = thrust::raw_pointer_cast(D.d_min_key.data());
  auto bi_ptr = thrust::raw_pointer_cast(D.d_min_idx.data());

  int threads = 256;
  int blocks = (num_vertices + threads - 1) / threads;
  size_t shared_mem =
      threads * (sizeof(weight_t) + sizeof(PrimGunrock::vertex_t));

  for (size_t i = 0; i < num_vertices; ++i) {
    // phase 1: block-level minima
    minKeyReduce<<<blocks, threads, shared_mem>>>(key_ptr, bk_ptr, bi_ptr,
                                                  num_vertices, INF);
    // phase 2: global min over block results
    minKeyReduce<<<1, threads, shared_mem>>>(bk_ptr, bk_ptr, bi_ptr, blocks,
                                             INF);

    // copy result
    weight_t minKey;
    int minIdx;
    hipMemcpy(&minKey, bk_ptr, sizeof(weight_t), hipMemcpyDeviceToHost);
    hipMemcpy(&minIdx, bi_ptr, sizeof(int), hipMemcpyDeviceToHost);
    if (minKey == INF)
      break;

    int u = minIdx;
    D.d_inMST[u] = 1;
    hipMemcpy(key_ptr + u, &INF, sizeof(weight_t), hipMemcpyHostToDevice);
    auto parent = D.d_parent[u];
    if (u != parent)
      mst_edges.emplace_back(parent, u, minKey);

    relaxEdges<<<1, 128>>>(row_ptr, col_ptr, w_ptr, key_ptr, p_ptr, in_ptr, u);
    hipDeviceSynchronize();
  }

  auto end = std::chrono::steady_clock::now();
  return std::chrono::duration_cast<std::chrono::seconds>(end - start);
}

Tree PrimGunrock::get_result() {
  // Copy parent array back to host
  std::vector<int> parent(num_vertices);
  thrust::copy(dev_->d_parent.begin(), dev_->d_parent.end(), parent.begin());

  // Compute total weight of the MST
  float totalWeight = 0;
  for (auto &edge : mst_edges)
    totalWeight += std::get<2>(edge);
  return Tree(num_vertices, parent, totalWeight);
}

} // namespace algos
