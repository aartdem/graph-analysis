#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>

#include "loader.hxx"
#include "prim.hxx"

namespace algos {

// Структура DeviceData хранит CSR-граф и вспомогательные буферы на GPU
struct PrimGunrock::DeviceData {
  using vertex_t = PrimGunrock::vertex_t;
  using edge_t = PrimGunrock::edge_t;
  using weight_t = PrimGunrock::weight_t;

  // CSR представление графа
  thrust::device_vector<edge_t> d_row_offsets;   // size = num_vertices+1
  thrust::device_vector<vertex_t> d_col_indices; // size = num_edges
  thrust::device_vector<weight_t> d_weight;      // size = num_edges

  // Хост-копия row_offsets для доступа из хоста
  thrust::host_vector<edge_t> h_row_offsets; // size = num_vertices+1

  // Буферы для алгоритма Прима
  thrust::device_vector<weight_t> d_key; // ключи (минимальные веса)
  thrust::device_vector<vertex_t> d_parent; // родители в MST
  thrust::device_vector<char> d_inMST; // флаги включения в MST (0/1)

  DeviceData() = default;
};

PrimGunrock::PrimGunrock()
    : num_vertices(0), num_edges(0), dev_(new DeviceData()) {}

PrimGunrock::~PrimGunrock() = default;

void PrimGunrock::load_graph(const std::filesystem::path &file_path) {
  using vertex_t = PrimGunrock::vertex_t;
  using edge_t = PrimGunrock::edge_t;
  using weight_t = PrimGunrock::weight_t;

  // 1) Загрузить исходный COO
  std::vector<vertex_t> coo_row, coo_col;
  std::vector<weight_t> coo_val;
  detail::load_mtx_coo<vertex_t, edge_t, weight_t>(file_path, coo_row, coo_col,
                                                   coo_val);

  // 2) Вычислить число вершин и исходных рёбер
  edge_t original_edges = static_cast<edge_t>(coo_row.size());
  vertex_t max_v = 0;
  for (vertex_t u : coo_row)
    max_v = std::max(max_v, u);
  for (vertex_t v : coo_col)
    max_v = std::max(max_v, v);
  num_vertices = max_v + 1;

  // 3) Построить удвоенный COO
  std::vector<vertex_t> srcs, dsts;
  std::vector<weight_t> weights;
  srcs.reserve(2 * original_edges);
  dsts.reserve(2 * original_edges);
  weights.reserve(2 * original_edges);

  for (edge_t i = 0; i < original_edges; ++i) {
    vertex_t u = coo_row[i], v = coo_col[i];
    weight_t w = coo_val[i];
    srcs.push_back(u);
    dsts.push_back(v);
    weights.push_back(w);
    if (u != v) {
      srcs.push_back(v);
      dsts.push_back(u);
      weights.push_back(w);
    }
  }
  num_edges = static_cast<edge_t>(srcs.size());

  // 4) Собрать локальный CSR:
  // 4.1) степени
  std::vector<vertex_t> degrees(num_vertices, 0);
  for (edge_t i = 0; i < num_edges; ++i) {
    degrees[srcs[i]]++;
  }

  // 4.2) эксклюзивный префикс-сумм
  std::vector<vertex_t> row_offsets_local(num_vertices + 1);
  row_offsets_local[0] = 0;
  for (vertex_t i = 0; i < num_vertices; ++i)
    row_offsets_local[i + 1] = row_offsets_local[i] + degrees[i];

  // 4.3) курсоры и fill
  std::vector<edge_t> cursor(row_offsets_local.begin(),
                             row_offsets_local.end());
  std::vector<vertex_t> col_idx_local(num_edges);
  std::vector<weight_t> w_local(num_edges);

  for (edge_t i = 0; i < num_edges; ++i) {
    vertex_t u = srcs[i];
    edge_t pos = cursor[u]++;
    col_idx_local[pos] = dsts[i];
    w_local[pos] = weights[i];
  }
  dev_->h_row_offsets = thrust::host_vector<vertex_t>(row_offsets_local.begin(),
                                                      row_offsets_local.end());

  // 5) Копируем локальные CSR в device_vector
  dev_->d_row_offsets = thrust::device_vector<vertex_t>(
      row_offsets_local.begin(), row_offsets_local.end());
  dev_->d_col_indices = thrust::device_vector<vertex_t>(col_idx_local.begin(),
                                                        col_idx_local.end());
  dev_->d_weight =
      thrust::device_vector<weight_t>(w_local.begin(), w_local.end());
}

std::chrono::seconds PrimGunrock::compute() {
  mst_edges.clear();
  using vertex_t = PrimGunrock::vertex_t;
  using edge_t = PrimGunrock::edge_t;
  using weight_t = PrimGunrock::weight_t;
  const weight_t INF = std::numeric_limits<weight_t>::max();

  auto start = std::chrono::steady_clock::now();
  if (num_vertices == 0)
    return {};

  auto &D = *dev_;
  // Ресайзим буферы
  D.d_key.resize(num_vertices);
  D.d_parent.resize(num_vertices);
  D.d_inMST.resize(num_vertices);

  thrust::fill(D.d_key.begin(), D.d_key.end(), INF);
  thrust::fill(D.d_parent.begin(), D.d_parent.end(), -1);
  thrust::fill(D.d_inMST.begin(), D.d_inMST.end(), 0);

  // Стартовая вершина = 0
  D.d_key[0] = 0;
  D.d_parent[0] = 0;

  // Сырые указатели для девайс-данных
  auto row_ptr = thrust::raw_pointer_cast(D.d_row_offsets.data());
  auto col_ptr = thrust::raw_pointer_cast(D.d_col_indices.data());
  auto w_ptr = thrust::raw_pointer_cast(D.d_weight.data());
  auto key_ptr = thrust::raw_pointer_cast(D.d_key.data());
  auto parent_ptr = thrust::raw_pointer_cast(D.d_parent.data());
  auto inMST_ptr = thrust::raw_pointer_cast(D.d_inMST.data());

  // Основной цикл Прима
  for (size_t iter = 0; iter < num_vertices; ++iter) {
    // 1) Выбираем вершину u с минимальным ключом
    auto it =
        thrust::min_element(thrust::device, D.d_key.begin(), D.d_key.end());
    vertex_t u = it - D.d_key.begin();
    weight_t minKey = *it;
    if (minKey == INF)
      break; // оставшиеся недостижимы

    // 2) Включаем u в MST
    thrust::fill_n(thrust::device, D.d_inMST.begin() + u, 1, (char)1);
    *it = INF; // исключаем из дальнейшего выбора
    if (u != 0) {
      // считываем именно parent[u] из device_vector
      vertex_t pu = D.d_parent[u];
      mst_edges.emplace_back(pu, u, minKey);
    }

    // 3) Параллельно обновляем ключи соседей
    edge_t e_start = D.h_row_offsets[u];
    edge_t e_end = D.h_row_offsets[u + 1];
    thrust::for_each(thrust::device,
                     thrust::make_counting_iterator<edge_t>(e_start),
                     thrust::make_counting_iterator<edge_t>(e_end),
                     [=] __device__(edge_t e) {
                       vertex_t v = col_ptr[e];
                       weight_t w = w_ptr[e];
                       if (!inMST_ptr[v] && w < key_ptr[v]) {
                         key_ptr[v] = w;
                         parent_ptr[v] = u;
                       }
                     });
  }

  auto end = std::chrono::steady_clock::now();
  return std::chrono::duration_cast<std::chrono::seconds>(end - start);
}

Tree PrimGunrock::get_result() {
  // Копируем родителей на хост
  std::vector<int> parent(num_vertices);
  thrust::copy(dev_->d_parent.begin(), dev_->d_parent.end(), parent.begin());
  // Считаем общий вес
  float totalW = 0;
  for (auto &t : mst_edges)
    totalW += std::get<2>(t);
  return Tree(num_vertices, parent, totalW);
}

} // namespace algos
